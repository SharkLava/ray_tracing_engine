#include "cuda_utils.h"
#include <stdio.h>

void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d: %s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}
